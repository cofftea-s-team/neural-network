#include "hip/hip_runtime.h"
#include "kernel.cuh"

namespace cuda_network {
	__global__
	void __dot_a_b_add_c_kernel(const double* _A, const double* _B, const double* _C, double* _D, size_t N, size_t M, size_t M1) {
		int i = blockIdx.x;
		int j = threadIdx.x;

		*(_D + i * M1 + j) = _C[j];
		for (int k = 0; k < M; ++k) {
			*(_D + i * M1 + j) += *(_A + i * M + k) * *(_B + k * M1 + j);
		}
	}

	void __dot_a_b_add_c(const double* a, const double* _B, const double* _C, double* d, size_t N, size_t M, size_t M1) {
		auto _A = cuda_alloc<double>(N * M);
		auto _D = cuda_alloc<double>(N * M1);

		hipMemcpy(_A, a, N * M * sizeof(double), hipMemcpyHostToDevice);

		__dot_a_b_add_c_kernel<<<N, M1>>>(_A, _B, _C, _D, N, M, M1); // applies dot on a, b and adds c
		hipDeviceSynchronize();

		hipMemcpy(d, _D, N * M1 * sizeof(double), hipMemcpyDeviceToHost); // copy result to host
		hipFree(_A);
		hipFree(_D);
	}

	__global__
	void __dot_a_transpose_b_kernel(const double* _A, const double* _B, double* _D, size_t N, size_t M, size_t M1) {
		int i = blockIdx.x;
		int j = threadIdx.x;

		*(_D + i * M1 + j) = 0;
		for (int k = 0; k < M; ++k) {
			*(_D + i * M1 + j) += *(_A + i * M + k) * *(_B + j * M + k);
		}
	}

	void __dot_a_transpose_b(const double* a, const double* _B, double* d, size_t N, size_t M, size_t M1) {
		auto _A = cuda_alloc<double>(N * M);
		auto _D = cuda_alloc<double>(N * M1);

		hipMemcpy(_A, a, N * M * sizeof(double), hipMemcpyHostToDevice);

		__dot_a_transpose_b_kernel<<<N, M1>>>(_A, _B, _D, N, M, M1); // applies dot on a, transposed b
		hipDeviceSynchronize();

		hipMemcpy(d, _D, N * M1 * sizeof(double), hipMemcpyDeviceToHost); // copy result to host
		hipFree(_A);
		hipFree(_D);
	}

	__global__
	void __dot_transpose_a_b_kernel(const double* _A, const double* _B, double* _D, size_t N, size_t M, size_t M1) {
		int i = blockIdx.x;
		int j = threadIdx.x;

		*(_D + i * M1 + j) = 0;
		for (int k = 0; k < M; ++k) {
			*(_D + i * M1 + j) += *(_A + k * N + i) * *(_B + k * M1 + j);
		}
	}

	void __dot_transpose_a_b(const double* a, const double* b, double* d, size_t N, size_t M, size_t M1) {
		auto _A = cuda_alloc<double>(N * M);
		auto _B = cuda_alloc<double>(M * M1);
		auto _D = cuda_alloc<double>(N * M1);

		hipMemcpy(_A, a, N * M * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(_B, b, M * M1 * sizeof(double), hipMemcpyHostToDevice);

		__dot_transpose_a_b_kernel<<<N, M1 >>>(_A, _B, _D, N, M, M1); // applies dot on transposed a, b
		hipDeviceSynchronize();

		hipMemcpy(d, _D, N * M1 * sizeof(double), hipMemcpyDeviceToHost); // copy result to host
		hipFree(_A);
		hipFree(_B);
		hipFree(_D);
	}

	__global__
	void __mul_a_b_kernel(double* _A, double* _B, size_t N, size_t M) {
		int i = blockIdx.x;
		int j = threadIdx.x;

		*(_A + i * M + j) *= *(_B + i * M + j);
	}

	void __mul_a_b(double* a, const double* b, size_t N, size_t M) {
		auto _A = cuda_alloc<double>(N * M);
		auto _B = cuda_alloc<double>(M * M);

		hipMemcpy(_A, a, N * M * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(_B, b, N * M * sizeof(double), hipMemcpyHostToDevice);

		__mul_a_b_kernel<<<N, M>>>(_A, _B, N, M); // applies dot on transposed a, b
		hipDeviceSynchronize();

		hipMemcpy(a, _A, N * M * sizeof(double), hipMemcpyDeviceToHost); // copy result to host
		hipFree(_A);
		hipFree(_B);
	}
}